#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#define length 4096
#define NUM_OF_THREADS 16384
#define NUM_OF_BITS 8 * sizeof(unsigned long long int)

__global__ void func(int* d_mtx_to_vec, unsigned long long int steps, unsigned long long int steps_remainder, int *d_L1_vector, int *d_L1_strategy, int iLonger, int iShorter){
	int temp[length], vect[NUM_OF_BITS], product, L1, logical, i, l;
	unsigned long long int number, index, iMax, iMin, iNumofZeros, iNum_temp;
	index = blockIdx.x * blockDim.x + threadIdx.x;

	iMax = (index + 1) *(steps) - 1;
	iMin = index * (steps);
	if(index < (steps_remainder) ) iMax += index + 1;
	else iMax += (steps_remainder);
	if(index <= (steps_remainder)) iMin += index;
	else iMin += (steps_remainder);
		 number = iMin;
		 for(l=0; l < iLonger; l++) {temp[l] = d_mtx_to_vec[l];}
		 product = 0;
			for(i = 1 ; iShorter > i; i++){
				iNum_temp = (unsigned long long int) 1 << i;
				iNumofZeros=(unsigned long long int) iNum_temp >> 1;
				
				logical = ((number+ iNumofZeros)/iNum_temp) % 2;
				vect[i] = (int) 2 * logical - 1;
					if(vect[i] > 0){for(l=0; l < iLonger; l++){temp[l] += d_mtx_to_vec[i * iLonger + l]; }}
					else {for(l=0; l < iLonger; l++){temp[l] -= d_mtx_to_vec[i * iLonger + l]; }}				
			}
			for(l= 0; l < iLonger; l++) {product += abs(temp[l]);}
			L1 = product;
			for(l=1; l<iShorter; l++){d_L1_strategy[index * (iShorter - 1) + l - 1] = vect[l];}

     for(number=iMin + 1; number <= iMax; number++){
		 product = 0;
			for(i = 1 ; iShorter > i; i++){
				iNum_temp = (unsigned long long int) 1 << i;
				iNumofZeros=(unsigned long long int) iNum_temp >> 1;
				if( ((number+ iNumofZeros) % iNum_temp) == 0 ) {vect[i]=-vect[i] ;					
					if(vect[i] > 0){for(l=0; l < iLonger; l++){temp[l] += 2 * d_mtx_to_vec[i * iLonger + l]; }}
					else {for(l=0; l < iLonger; l++){temp[l] -= 2 *d_mtx_to_vec[i * iLonger + l]; }}
				break;
				}
            		}
	     for(l = 0; l < (iLonger ); l++) {product += abs(temp[l]);}
	     if(product > L1) {L1 = product;
		for(l=1; l<iShorter; l++){d_L1_strategy[index * (iShorter - 1) + l - 1] = vect[l];}
		}
     }
d_L1_vector[index] = L1;
}

int** mtx_read(int *iRows, int *iCols, char* fileName){
	int i = 0,j = 0, k = 0;
	int *row, **mtx, value;
	
	mtx = NULL;
	row = NULL;
	
	char g, cNum[256];
	
	FILE *fp;
	fp = fopen(fileName,"r");
	
	do{
		g = fgetc(fp);	
		if((((g - '0') < 10) && ((g - '0') >= 0)) || (g == 'e') || ( g == 'E') || (g == '.') || (g == '+') || (g == '-')) {cNum[i] = g; i++;}
		else {
			cNum[i] = '\0'; 
			if(cNum[0] != '\0') {sscanf(cNum, "%d", &value); j++; i = 0;  row = (int*) realloc(row, j * sizeof(int)); row[j-1] = value;}
			if( ((g == '\n') || (g == EOF)) && (j > 0)){*iCols = j; j = 0; k++; mtx = (int**) realloc(mtx, k * sizeof(int*)); mtx[k-1] = row; row = NULL;}
		}
		
	}while(!feof(fp));
	*iRows = k;
printf("rows: %d, cols: %d\n",*iRows, *iCols); 
	fclose(fp);
return mtx;
}

void fileN(char *fileName, char** argv, int *argc){
	int r;
	FILE *fp;
	if((*argc) < 2) {
		do{
			printf("Please give me a filename: "); 
			r = scanf("%s",fileName);
		}while(r != 1);
	}
	else sprintf(fileName,"%s", argv[1]);

	fp = fopen(fileName, "r");
	if(fp == NULL) {
		do{
			printf("Please give me a filename that exist within this directory: ");
			r = scanf("%s",fileName);
			fp = fopen(fileName, "r");
		}while(fp == NULL);
	}
	fclose(fp);
}

void calc_Lnorm(int* iRows, int* iCols, int* L1_max, int** mtx){
	int i, j/*, iMax*/, *mtx_to_vec, *d_mtx_to_vec, iShorter, iLonger, *L1_vector, *d_L1_vector, *L1_strategy, *d_L1_strategy, num_ofBlock, num_ofThread;
	unsigned long long int steps, steps_remainder, Inner_num , copyNum;
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	mtx_to_vec = (int*)calloc( *iRows * *iCols, sizeof(int));
	if( *iRows > *iCols ){
		for(j = 0; j < *iCols; j++){
			for(i = 0; i < *iRows; i++){
				mtx_to_vec[j * *iRows + i] = mtx[i][j];
			}
		}
	}
	else{
		for(i = 0; i < *iRows; i++){
			for(j = 0; j < *iCols; j++){
				mtx_to_vec[i * *iCols + j] = mtx[i][j];
			}
		}
	}
	if(*iRows < *iCols) {iShorter = *iRows; iLonger = *iCols;}
	else {iShorter = *iCols; iLonger = *iRows;}
	if(iShorter > (NUM_OF_BITS)) {printf("Matrix is too big. The number of rows or columns can not be more than %lu.\n", NUM_OF_BITS); exit(-1);}
	if(iLonger > length) {printf("Matrix is too big. The length variable %d should be bigger or equal than %d.\n", length, iLonger); exit(-1);}
	hipMalloc((void**)&d_mtx_to_vec, iShorter * iLonger * sizeof(int));
	Inner_num = (unsigned long long int) 1 << (iShorter - 1);
	copyNum = NUM_OF_THREADS > Inner_num ? Inner_num : NUM_OF_THREADS;
	num_ofThread = copyNum < devProp.warpSize ? copyNum : devProp.warpSize;
	num_ofBlock = copyNum/num_ofThread; copyNum = num_ofBlock * num_ofThread;
	if((NUM_OF_THREADS % num_ofThread) != 0) {printf("The NUM_OF_THREADS variable must be divisible with the number of threads in one block which is %d. Please modify the NUM_OF_THREADS variable and recompile this code again.\n", num_ofThread); exit(-1);}
	steps=Inner_num/copyNum; steps_remainder = Inner_num % copyNum;
	L1_vector = (int*) malloc(copyNum * sizeof(int));
	L1_strategy = (int*) malloc(copyNum * (iShorter - 1) * sizeof(int));
	hipMalloc((void**)&d_L1_vector, copyNum * sizeof(int));
	hipMalloc((void**)&d_L1_strategy, copyNum * (iShorter - 1) * sizeof(int));
	hipMemcpy(d_mtx_to_vec, mtx_to_vec, iShorter * iLonger * sizeof(int), hipMemcpyHostToDevice);
//	printf("num_ofBlock: %d, num_ofThread: %d\n",num_ofBlock,num_ofThread);
	func<<<num_ofBlock,num_ofThread>>>(d_mtx_to_vec, steps, steps_remainder, d_L1_vector, d_L1_strategy, iLonger, iShorter);
	hipMemcpy(L1_vector, d_L1_vector, copyNum * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(L1_strategy, d_L1_strategy, copyNum * (iShorter - 1) * sizeof(int), hipMemcpyDeviceToHost);
	*L1_max = L1_vector[0]; //iMax = 0;
	for(i = 1; i < copyNum; i++){ if(*L1_max < L1_vector[i]) {*L1_max = L1_vector[i]; /*iMax = i;*/}}

/*	printf("L1 is: %d\n",*L1_max);

	FILE *fp;
	fp = fopen("strategy_L1.txt", "w");
	fprintf(fp,"1\n");
	for(i=0; i<(iShorter - 1); i++) {fprintf(fp, "%d\n", L1_strategy[iMax * (iShorter - 1) + i]);}
	fclose(fp);
*/
	free(L1_vector);
	free(L1_strategy);
	free(mtx_to_vec);
	hipFree(d_L1_vector);
	hipFree(d_L1_strategy);
	hipFree(d_mtx_to_vec);
}

void partitionStrategy(int *iPartition, int* iRows, int* iRows0, int* iRows1, int* iCols, int* iShorter0, int* iShorter1, int* iLonger0, int* iLonger1){
	int i;
	*iRows0 = 0;
	*iRows1 = 0;
	for(i = 0; i < (*iRows); i++){
		iPartition[i] = rand() % 2;
		if(iPartition[i] == 0) {(*iRows0)++;}
		else {(*iRows1)++;}
	}

	if((*iRows0) < (*iCols)) {(*iShorter0) = (*iRows0); (*iLonger0) = (*iCols);}
	else {(*iShorter0) = (*iCols); (*iLonger0) = (*iRows0);}

	if((*iRows1) < (*iCols)) {(*iShorter1) = (*iRows1); (*iLonger1) = (*iCols);}
	else {(*iShorter1) = (*iCols); (*iLonger1) = (*iRows1);}
}

void partition_matrix(int** mtx, int** mtx0, int** mtx1, int* iPartition, int* iRows, int* iCols, int* iRows0, int* iRows1){
	int i, i0 = 0, i1 = 0, j;
	for(i = 0; i < (*iRows); i++){
		if(iPartition[i] == 0) {
				for(j = 0; j < (*iCols); j++) {
					mtx0[i0][j] = mtx[i][j];
					//printf("mtx: %d, mtov: %d\n", mtx[i][j], mtx_to_vec0[i * (*iCols) + j]);
				}
		i0++;
		}
		else {
				for(j = 0; j < (*iCols); j++) {
					mtx1[i1][j] = mtx[i][j];
					//printf("mtx: %d, mtov: %d\n", mtx[i][j], mtx_to_vec1[i * (*iCols) + j]);
				}
		i1++;
		}
	}
}

void mtx_free(int* iRows, int** mtx){
	int i;
	for(i = 0; i < *iRows; i++){
		free(mtx[i]);
	}
	free(mtx);
}

int** mtx_allocate(int* iRows, int* iCols){
	int i, **mtx;
	mtx = (int**) calloc(*iRows, sizeof(int*));
	for(i = 0; i < *iRows; i++){
		mtx[i] = (int*) calloc(*iCols, sizeof(int));
	}
return mtx;
}

int main(int argc, char *argv[]){
	char fileName[1024];
	int i, iRows, iCols, iShorter0, iShorter1, iLonger0, iLonger1, *iPartition, **mtx, **mtx0, **mtx1, iRows0, iRows1, S_max = 0, L10, L11;
	FILE *partition_strategy;
	fileN(fileName, argv, &argc);
	mtx = mtx_read(&iRows, &iCols, fileName);
	srand(time(NULL));

	iPartition = (int*) calloc(iRows, sizeof(int));
	while(1){
		partitionStrategy(iPartition, &iRows, &iRows0, &iRows1, &iCols, &iShorter0, &iShorter1, &iLonger0, &iLonger1);
		if( (iShorter0 < 41) && (iShorter1 < 41) ){
		mtx0 = mtx_allocate(&iRows0, &iCols);
		mtx1 = mtx_allocate(&iRows1, &iCols);
		partition_matrix(mtx, mtx0, mtx1, iPartition, &iRows, &iCols, &iRows0, &iRows1);
		calc_Lnorm(&iRows0, &iCols, &L10, mtx0);
		calc_Lnorm(&iRows1, &iCols, &L11, mtx1);
if((L10 + L11) > S_max ){
	S_max = L10 + L11;

	partition_strategy = fopen("Partition_strategy.txt", "a");
	fprintf(partition_strategy, "L1_0: %d, L1_1: %d, S_max: %d\n",L10, L11, S_max);
	for(i=0; i < iRows; i++){
		fprintf(partition_strategy, "%d\t", iPartition[i]);
	}
	fprintf(partition_strategy, "\n");
fclose(partition_strategy);
}
	mtx_free(&iRows1, mtx1);
	mtx_free(&iRows0, mtx0);
	}
	}
free(iPartition);
mtx_free(&iRows, mtx);
return 0;  
}
